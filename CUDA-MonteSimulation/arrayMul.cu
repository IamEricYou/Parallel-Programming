#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void ArrayMul( float *A, float *B, float *C )
{
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	prods[tnum] = A[gid] * B[gid];

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = prods[0];
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

	float * hA = new float [ SIZE ];
	float * hB = new float [ SIZE ];
	float * hC = new float [ SIZE/BLOCKSIZE ];

	for( int i = 0; i < SIZE; i++ )
	{
		hA[i] = hB[i] = (float) sqrt(  (float)(i+1)  );
	}

	// allocate device memory:

	float *dA, *dB, *dC;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), (SIZE/BLOCKSIZE)*sizeof(float) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dA, hA, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, hB, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
	        ArrayMul<<< grid, threads >>>( dA, dB, dC );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaMultReductions/Second = %10.2lf\n", SIZE, megaMultsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hC, dC, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	// check the sum :

	double sum = 0.;
	for(int i = 0; i < SIZE/BLOCKSIZE; i++ )
	{
		//fprintf(stderr, "hC[%6d] = %10.2f\n", i, hC[i]);
		sum += (double)hC[i];
	}
	fprintf( stderr, "\nsum = %10.2lf\n", sum );

	// clean up memory:
	delete [ ] hA;
	delete [ ] hB;
	delete [ ] hC;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );


	return 0;
}

